#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "transpose_impl.h"

namespace onnxruntime {
namespace cuda {

constexpr int TILE_DIM = 32;
constexpr int BLOCK_ROWS = 8;

template <typename T>
__global__ void _Transpose3DKernel(const TArray<int64_t> input_shape,
                                   const TArray<int64_t> input_strides,
                                   const T* input_data, T* output_data) {
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = input_shape[2];

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    tile[threadIdx.y+j][threadIdx.x] = input_data[blockIdx.z * input_strides[0] + (y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    output_data[blockIdx.z * input_strides[0] + (y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

bool canDoTranspose3D(int32_t rank,
                      const std::vector<int64_t>& input_dims,
                      const std::vector<size_t>& permutations) {
  if (rank == 3 &&
      input_dims[1] % TILE_DIM == 0 && input_dims[2] % TILE_DIM == 0 &&
      permutations[1] == 2 && permutations[2] == 1) {
    return true;
  }
  return false;
}

Status Transpose3DImpl(size_t element_size,
                       const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
                       const void* input_data, void* output_data, int64_t N) {
  dim3 block_size(TILE_DIM, BLOCK_ROWS);
  dim3 grid_size(input_shape[2]/TILE_DIM, input_shape[1]/TILE_DIM, input_shape[0]);

  switch (element_size) {
    case sizeof(int8_t):
      _Transpose3DKernel<int8_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data));
      break;
    case sizeof(int16_t):
      _Transpose3DKernel<int16_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const int16_t*>(input_data),
          reinterpret_cast<int16_t*>(output_data));
      break;
    case sizeof(int32_t):
      _Transpose3DKernel<int32_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data));
      break;
    case sizeof(int64_t):
      _Transpose3DKernel<int64_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data));
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                              element_size);
  }

  return Status::OK();
}

template <typename T, int element_size>
__global__ void _Transpose4DKernel(const TArray<int64_t> input_strides, const T* input_data,
                                   const TArray<int64_t> output_strides, T* output_data, CUDA_LONG N) {
  // output coordinates will be: blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x
  CUDA_LONG input_index = (blockIdx.y * input_strides[0] +
                           blockIdx.x * input_strides[1] +
                           threadIdx.y * input_strides[2]) / (4 * 4 / element_size) +
                           threadIdx.x * input_strides[3];

  CUDA_LONG output_index = (blockIdx.y * output_strides[0] +
                            blockIdx.x * output_strides[1] +
                            threadIdx.y * output_strides[2]) / (4 * 4 / element_size) +
                            threadIdx.x * output_strides[3];

  const int4* v_input = reinterpret_cast<const int4*>(input_data);
  int4* v_output = reinterpret_cast<int4*>(output_data);

  if (input_index < N && output_index < N) {
    v_output[output_index] = v_input[input_index];
  }
}

bool canDoTranspose4D(const hipDeviceProp_t& prop,
                      size_t element_size,
                      int32_t rank,
                      const std::vector<int64_t>& input_dims,
                      const std::vector<size_t>& permutations) {
  // In Tranpose4D, vector of int4 will be used.
  int64_t num_threads_per_block = input_dims[2] * input_dims[3] / (4 * sizeof(int) / element_size);
  if (rank == 4 && permutations[3] == 3 &&
      num_threads_per_block <= prop.maxThreadsPerBlock &&
      // num_threads_per_block must be aligned with warp size: 32
      ((num_threads_per_block & (prop.warpSize - 1)) == 0)) {
    return true;
  }
  return false;
}

Status Transpose4DImpl(size_t element_size, const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides, const void* input_data,
                       const TArray<int64_t>& output_strides, void* output_data, int64_t N) {
  dim3 block_size(input_shape[3], input_shape[2]);
  dim3 grid_size(input_shape[1], input_shape[0]);

  switch (element_size) {
    case sizeof(int8_t):
      block_size.x = block_size.x / (4 * 4 / sizeof(int8_t));
      N /= (4 * 4 / sizeof(int8_t));
      _Transpose4DKernel<int8_t, sizeof(int8_t)><<<grid_size, block_size, 0>>>(
          input_strides, reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          output_strides, reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data), N);
      break;
    case sizeof(int16_t):
      block_size.x = block_size.x / (4 * 4 / sizeof(int16_t));
      N /= (4 * 4 / sizeof(int16_t));
      _Transpose4DKernel<int16_t, sizeof(int16_t)><<<grid_size, block_size, 0>>>(
          input_strides, reinterpret_cast<const int16_t*>(input_data),
          output_strides, reinterpret_cast<int16_t*>(output_data), N);
      break;
    case sizeof(int32_t):
      block_size.x = block_size.x / (4 * 4 / sizeof(int32_t));
      N /= (4 * 4 / sizeof(int32_t));
      _Transpose4DKernel<int32_t, sizeof(int32_t)><<<grid_size, block_size, 0>>>(
          input_strides, reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          output_strides, reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data), N);
      break;
    case sizeof(int64_t):
      block_size.x = block_size.x / (4 * 4 / sizeof(int64_t));
      N /= (4 * 4 / sizeof(int64_t));
      _Transpose4DKernel<int64_t, sizeof(int64_t)><<<grid_size, block_size, 0>>>(
          input_strides, reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          output_strides, reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data), N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                              element_size);
  }

  return Status::OK();
}

template <typename T>
__global__ void _TransposeKernel(int32_t shape_rank, const TArray<int64_t> input_strides,
                                 const T* input_data, const TArray<fast_divmod> output_strides, T* output_data, CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

  #pragma unroll
  for (auto dim = 0; dim < input_strides.GetCapacity(); ++dim) {
    if (dim >= shape_rank) {
      break;
    }
    int out_coord, r;
    output_strides[dim].divmod(output_index, out_coord, r);
    output_index = r;
    input_index += input_strides[dim] * out_coord;
  }
  output_data[id] = input_data[input_index];
}

Status TransposeImpl(size_t element_size, int32_t shape_rank, const TArray<int64_t>& input_strides,
                     const void* input_data, const TArray<fast_divmod>& fdm_output_strides, void* output_data, int64_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  switch (element_size) {
    case sizeof(int8_t):
      _TransposeKernel<int8_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int16_t):
      _TransposeKernel<int16_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int32_t):
      _TransposeKernel<int32_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int64_t):
      _TransposeKernel<int64_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data),
          N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
